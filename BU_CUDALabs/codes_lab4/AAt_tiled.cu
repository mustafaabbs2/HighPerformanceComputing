#include "hip/hip_runtime.h"
// A*At = B
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <sys/time.h>

#define BLOCKSIZE 4

typedef struct
{	int width;
	int height;
	float *elements;
} Matrix;

double get_time() 
{  struct timeval tim;
  hipDeviceSynchronize();
  gettimeofday(&tim, NULL);
  return (double) tim.tv_sec+(tim.tv_usec/1000000.0);
}

__global__ void transpose(Matrix A, Matrix B, int W, int H, int MAX)
{
	int i = threadIdx.x;
	int j = threadIdx.y;

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int I = bx*BLOCKSIZE + i;	
	int J = by*BLOCKSIZE + j;	

	__shared__ float 	block[BLOCKSIZE][BLOCKSIZE],
				block_t[BLOCKSIZE][BLOCKSIZE];

	float sum;
	sum = 0.0f;
	for(int m=0; m<MAX/BLOCKSIZE; m++)
	{	block[i][j] = A.elements[J*MAX + m*BLOCKSIZE + i];

		block_t[i][j] = A.elements[bx*BLOCKSIZE*MAX + i*MAX + m*BLOCKSIZE + j]; // Need to transpose within each block and the blocks themselves
		//block_t[j][i] = A.elements[bx*BLOCKSIZE*MAX + j*MAX + m*BLOCKSIZE + i]; // Equivalent to above and helps memory coalescing

		__syncthreads();

		for (int k=0; k<BLOCKSIZE; k++)
			sum += block[k][j]*block_t[i][k];

		__syncthreads();

	}
	B.elements[J*MAX + I] = sum;

}


// Host code
int main()
{
	hipSetDevice(1);
	int W = 16; // matrix width
	int H = 16; // matrix height

	int W_max = ((W-0.5)/BLOCKSIZE + 1)*BLOCKSIZE;
	int H_max = ((H-0.5)/BLOCKSIZE + 1)*BLOCKSIZE;

	int MAX;
	if (W_max/H_max == 0){MAX = H_max;}
	else MAX = W_max;

	Matrix A, B;
	
	A.width = W;
	B.width = H;
	A.height = H;
	B.height = W;

	A.elements = new float [MAX*MAX];
	B.elements = new float [MAX*MAX];

	int size = MAX*MAX*sizeof(float);

	// Initialize matrix
	for (int j=0; j<MAX; j++)
	{	for (int i=0; i<MAX; i++)
		{	A.elements[j*MAX+i] = 0.0f;
		}
	}

	// Fill up matrix
        std::ifstream A_input;
        A_input.open("A.txt");

        float a;
        A_input >> a;
        while (!A_input.eof())  
        {       for (int j=0; j<H; j++)
                {       for (int i=0; i<W; i++)
                        {       A.elements[j*MAX+i] = a;
                                A_input >> a;
                        }   
                }   
        }   
        A_input.close();


 	// Allocate in GPU
	Matrix A_d, B_d;

	A_d.width = W;
	B_d.width = H;
	A_d.height = H;
	B_d.height = W;

	hipMalloc( (void**) &A_d.elements, size);
	hipMalloc( (void**) &B_d.elements, size);

	hipMemcpy(A_d.elements, A.elements, size, hipMemcpyHostToDevice);

	dim3 dimBlock (BLOCKSIZE, BLOCKSIZE);
	dim3 dimGrid (MAX/BLOCKSIZE, MAX/BLOCKSIZE);
	double start = get_time();
	transpose<<<dimGrid, dimBlock>>>(A_d, B_d, W, H, MAX);
	double stop = get_time();

	double time = stop-start;

	hipMemcpy(B.elements, B_d.elements, size, hipMemcpyDeviceToHost);

	// Print results
	for (int j=0; j<H; j++)
	{	for (int i=0; i<H; i++)
			std::cout<<B.elements[j*MAX + i]<<"\t";
		std::cout<<std::endl;
	}

	std::cout<<time<<std::endl;
	hipFree(A_d.elements);	
	hipFree(B_d.elements);	
	
}
