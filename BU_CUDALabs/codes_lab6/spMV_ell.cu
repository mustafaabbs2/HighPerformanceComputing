#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/array1d.h>
#include <cusp/ell_matrix.h>
#include <cusp/gallery/poisson.h>
#include <cusp/print.h>
#include <cusp/multiply.h>
#include <cusp/transpose.h>
#include <thrust/device_ptr.h>

#define BSZ 2

__global__ void ell_matvec(float *elem, int *col, float *b, float *c, int num_entries, int N_side)
{
	// index i runs through the rows of the matrix
	int i = blockIdx.x * BSZ + threadIdx.x;

	if (i>=N_side) {return;}
	
	float sum = 0.0f;
	int column;
	for (int j=0; j<num_entries; j++)
	{	column = col[j*N_side + i];
		if (column!=-1)
		sum += elem[j*N_side + i] * b[column];
	}
	c[i] = sum;
	
}


int main()
{	
	int N = 4;
	int N_side = N*N;

	// Allocate and create matrix
	cusp::ell_matrix <int, float, cusp::device_memory> A;
	cusp::gallery::poisson5pt(A, N, N);

	int ell_col = A.values.num_cols;
	int ell_row = A.values.num_rows;

	cusp::array1d <float, cusp::device_memory> A_val(ell_row*ell_col);
	cusp::array1d <float, cusp::device_memory> A_col(ell_row*ell_col);

	// Allocate and create vector
	cusp::array1d <float, cusp::device_memory> b(N_side, 1.0f);

        float *c_d;
        hipMalloc( (void**) &c_d, N_side*sizeof(float));
        thrust::device_ptr<float> c_ptr(c_d);

	// Recover array pointers
	// In order to get raw_pointer_cast to recover de data
	// in a sensible way, I need to transpose before, which gives me
	// a column major order. Recovering the non transpose array2d
	// does not give a row major order, but something else
	cusp::array2d <float, cusp::device_memory> At_val;
	cusp::transpose(A.values, At_val);
	cusp::array2d <int, cusp::device_memory> At_col;
	cusp::transpose(A.column_indices, At_col);

	float *val_d = thrust::raw_pointer_cast(&At_val(0,0));
	int   *col_d = thrust::raw_pointer_cast(&At_col(0,0));
	float *b_d   = thrust::raw_pointer_cast(&b[0]);

	//thrust::device_ptr<float> val_ptr(val_d);

	int dimGrid (int((N_side-0.5)/BSZ) + 1);
	int dimBlock (BSZ);

	ell_matvec <<<dimGrid, dimBlock>>> (val_d, col_d, b_d, c_d, ell_col, N_side);

	cusp::array1d<float, cusp::device_memory> c(c_ptr, c_ptr+N_side);

	cusp::print(c);

	return 0;

}
