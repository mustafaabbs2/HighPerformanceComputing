
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU! \n");
}

int main()
{   printf("Hello World from CPU! \n");
    helloFromGPU <<<1,10>>>();
    hipDeviceReset();
    return 0;
}