//C headers
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"


#include <hipblas.h>
#include <math.h>
#include <assert.h>

//initialize a vector of size n with random values  

void vector_init(float *v, int n)
{
    for (int i = 0; i < n; i++)
    {
        v[i] = rand() / (float)RAND_MAX;
    }
}

//verify the result

void verify_result(float *a, float *b, float *c, int n)
{
    for (int i = 0; i < n; i++)
    {
        assert(fabs(a[i] + b[i] - c[i]) < 1e-3);
    }
}

int main()
{
    int n = 1<<2; //size of the vector
    float *h_a, *h_b, *h_c; //vectors a, b, c
    float *d_a, *d_b; //device vectors a, b

//Allocate memory
    h_a = (float *)malloc(n * sizeof(float));
    h_b = (float *)malloc(n * sizeof(float));
    h_c = (float *)malloc(n * sizeof(float));

    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_b, n * sizeof(float));

    vector_init(h_a, n);
    vector_init(h_b, n);
    

//create and initialize the cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    //copy the host vectors to the device

    hipblasSetVector(n, sizeof(float), h_a, 1, d_a, 1); //the last element is the step size
    hipblasSetVector(n, sizeof(float), h_b, 1, d_b, 1); //the last element is the step size

    const float scale = 1.0f;
    hipblasSaxpy(handle, n, &scale, d_a, 1, d_b, 1); //ax+b, set a to 1.0f

    hipblasGetVector(n, sizeof(float), d_b, 1, h_c, 1);

    verify_result(h_a, h_b, h_c, n);

    hipblasDestroy(handle);


    hipFree(d_a);
    hipFree(d_b);
    free(h_a);
    free(h_b);

    return 0;
// nvcc -o 5_3_cublasvectoradd -lcublas 5_3_cublasvectoradd.cu       
}


