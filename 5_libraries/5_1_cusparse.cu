#include <iostream>
#include <thrust/version.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

int main(void)
{
    int cuda_major = CUDART_VERSION / 1000;
    int cuda_minor = (CUDART_VERSION % 1000) / 10;
    int thrust_major = THRUST_MAJOR_VERSION;
    int thrust_minor = THRUST_MINOR_VERSION;

    hipsparseHandle_t handle;
    hipsparseCreate(&handle);

    // Get cusparse version information
    int cusparse_version;
    hipsparseGetVersion(handle, &cusparse_version);
    int cusparse_major = cusparse_version / 1000;
    int cusparse_minor = (cusparse_version % 1000) / 10;

    hipsparseDestroy(handle);

    std::cout << "CUDA       v" << cuda_major << "." << cuda_minor << std::endl;
    std::cout << "Thrust     v" << thrust_major << "." << thrust_minor << std::endl;
    std::cout << "cusparse   v" << cusparse_major << "." << cusparse_minor << std::endl;

    return 0;
}
