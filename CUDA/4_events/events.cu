
#include <hip/hip_runtime.h>

#include <algorithm>
#include <cassert>
#include <chrono>
#include <cstdlib>
#include <ctime>
#include <iostream>
#include <numeric>
#include <stdio.h>
#include <vector>

__global__ void helloFromGPU(void)
{
	printf("Hello World from GPU! \n");
}

__global__ void addStreams(int* in, int* out, int size)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if(gid < size)
	{
		for(int i = 0; i < 25; i++)
		{
			out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
		}
	}
}

//Wrapper Functions:

void helloWorldEvents_()
{
	printf("Hello World from CPU! \n");

	float et;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	//Start timing
	hipEventRecord(start);

	helloFromGPU<<<1, 10>>>();

	//Synchronize
	hipDeviceSynchronize();

	//Stop
	hipEventRecord(stop);

	//Sync events
	hipEventSynchronize(stop);

	//Calculate et = elapsed time
	hipEventElapsedTime(&et, start, stop);

	printf("The elapsed time is  %f milliseconds", et);

	hipDeviceReset();
}

void helloWorldStreams_()
{
	int dev = 0;
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, dev);

	if(deviceProp.concurrentKernels == 0)
	{
		printf("> GPU does not support concurrent kernel execution \n");
		printf("kernel execution will be serialized \n");
	}

	hipStream_t str1, str2, str3;

	hipStreamCreate(&str1);
	hipStreamCreate(&str2);
	hipStreamCreate(&str3);

	helloFromGPU<<<1, 1, 0, str1>>>();
	helloFromGPU<<<1, 1, 0, str2>>>();
	helloFromGPU<<<1, 1, 0, str3>>>();

	hipStreamDestroy(str1);
	hipStreamDestroy(str2);
	hipStreamDestroy(str3);

	hipDeviceSynchronize();
	hipDeviceReset();
}

void addStreams_()
{
	int size = 1 << 18;
	int byte_size = size * sizeof(int);

	//initialize host pointer
	int *h_in, *h_ref, *h_in2, *h_ref2;

	hipHostMalloc((void**)&h_in, byte_size, hipHostMallocDefault);
	hipHostMalloc((void**)&h_ref, byte_size, hipHostMallocDefault);
	hipHostMalloc((void**)&h_in2, byte_size, hipHostMallocDefault);
	hipHostMalloc((void**)&h_ref2, byte_size, hipHostMallocDefault);

	//allocate device pointers
	int *d_in, *d_out, *d_in2, *d_out2;
	hipMalloc((void**)&d_in, byte_size);
	hipMalloc((void**)&d_out, byte_size);
	hipMalloc((void**)&d_in2, byte_size);
	hipMalloc((void**)&d_out2, byte_size);

	hipStream_t str, str2;
	hipStreamCreate(&str);
	hipStreamCreate(&str2);

	//kernel launch
	dim3 block(128);
	dim3 grid(size / block.x);

	//transfer data from host to device
	hipMemcpyAsync(d_in, h_in, byte_size, hipMemcpyHostToDevice, str);
	addStreams<<<grid, block, 0, str>>>(d_in, d_out, size);
	hipMemcpyAsync(h_ref, d_out, byte_size, hipMemcpyDeviceToHost, str);

	hipMemcpyAsync(d_in2, h_in2, byte_size, hipMemcpyHostToDevice, str2);
	addStreams<<<grid, block, 0, str2>>>(d_in2, d_out2, size);
	hipMemcpyAsync(h_ref2, d_out2, byte_size, hipMemcpyDeviceToHost, str2);

	hipStreamSynchronize(str);
	hipStreamDestroy(str);

	hipStreamSynchronize(str2);
	hipStreamDestroy(str2);

	hipDeviceReset();
}
