// Matrix multiplication by parts
// Elements stored in row-major order

using namespace std;

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <fstream>
#define BLOCK_SIZE 16

typedef struct
{	int width;
	int height;
	float *elements;
} Matrix;

// Forward declaration of matrix mult
__global__ void MatMulKernel (const Matrix, const Matrix, Matrix);

// Host code
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
	// Load matrices A and B to device memory
	Matrix d_A;
	d_A.width = A.width; d_A.height = A.height;
	size_t size = A.width * A.height * sizeof(float);
	hipMalloc((void**) &d_A.elements, size);
	hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);
	
	Matrix d_B;
	d_B.width = B.width; d_B.height = B.height;
	size = B.width * B.height * sizeof(float);
	hipMalloc((void**) &d_B.elements, size);
	hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);
	
	// allocate C in device
	Matrix d_C;
	d_C.width = C.width; d_C.height = C.height;
	size = d_C.width * d_C.height * sizeof(float);
	hipMalloc((void**) &d_C.elements, size);
	
	// call kernel
        dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
        dim3 dimGrid(int(B.width-0.5)/BLOCK_SIZE+1, int(A.height-0.5)/BLOCK_SIZE+1);
	MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);
	
	// copy C to host
	hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);
	
	// free device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);
}

//matrix multiplication kernel
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{	
	// each thread computes one element of C and acumulates results to Cvalue
        float Cvalue = 0;
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
	
	if ((row>=A.height) || (col>=B.width)){return;}    

        for (int e=0; e<A.width; e++)
                Cvalue += A.elements[row*A.width + e] * B.elements[e*B.width + col];
        C.elements[row*C.width + col] = Cvalue;

}

int main(int argc, char * const argv[])
{	
	int Width = 16;
	
	Matrix A;
	Matrix B;
	Matrix C;
	
	A.width = Width;
	B.width = Width;
	C.width = Width;
	
	A.height = Width;
	B.height = Width;
	C.height = Width;
	
	A.elements = new float[Width*Width];
	B.elements = new float[Width*Width];
	C.elements = new float[Width*Width];
	
	//fill matrices
	std::ifstream A_input;
	std::ifstream B_input;
	A_input.open("A.txt");
	B_input.open("B.txt");
	
	float a, b;
	A_input >> a;	
	B_input >> b;	
	int i = 0;
	while (!A_input.eof())
	{	A.elements[i] = a;
		B.elements[i] = b;
		A_input >> a;	
		B_input >> b;	
		i += 1;
	}
	A_input.close();
	B_input.close();

	MatMul(A, B, C);
	std::ofstream C_output;
	C_output.open("C.txt");
	for (int i=0; i<Width; i++)
	{	for (int j=0; j<Width; j++)
			C_output<<C.elements[i*Width+j]<<"\t";
		C_output<<endl;
	}

}
	
