#include "hip/hip_runtime.h"
#include <iostream>

#include <cusp/array1d.h>
#include <cusp/csr_matrix.h>
#include <cusp/gallery/poisson.h>
#include <cusp/print.h>
#include <cusp/multiply.h>
#include <thrust/device_ptr.h>

#define BSZ 4

__global__ void csr_matvec(float *elem, int *row_off, int *col, float *b, float *c, int N_side)
{
	// Write your kernel here!
}


int main()
{	
	int N = 4;
	int N_side = N*N;
	//int N_el = 5*N_side-4*N;

	// Allocate and create matrix (the poisson5pt matrix has 5*N*N-4*N elements)
	cusp::csr_matrix <int, float, cusp::device_memory> A;
	cusp::gallery::poisson5pt(A, N, N);

	// Allocate and create vector
	cusp::array1d <float, cusp::device_memory> b(N_side, 1.0f);

	//cusp::array1d <float, cusp::device_memory> c(N_side);
	float *c_d;
	hipMalloc( (void**) &c_d, N_side*sizeof(float));
	thrust::device_ptr<float> c_ptr(c_d);

	// Recover array pointers
	float *elements = thrust::raw_pointer_cast(&A.values[0]);
	int   *rows_off = thrust::raw_pointer_cast(&A.row_offsets[0]);
	int   *columns 	= thrust::raw_pointer_cast(&A.column_indices[0]);
	float *b_d	= thrust::raw_pointer_cast(&b[0]);

	int dimGrid (int((N_side-0.5)/BSZ) + 1);
	int dimBlock (BSZ);

	csr_matvec <<<dimGrid, dimBlock>>> (elements, rows_off, columns, b_d, c_d, N_side);

	cusp::array1d<float, cusp::device_memory> c(c_ptr, c_ptr+N_side);	

	cusp::print(c);

	return 0;

}
