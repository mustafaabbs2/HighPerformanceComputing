#include "hip/hip_runtime.h"
//C headers
#include <stdio.h>
#include <stdlib.h>
//need these CUDA headers
#include "hip/hip_runtime.h"
#include ""
//Contains self written helper functions
#include "../common/common.h"


__global__ void without_divergence()
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	float a, b;
	a = b = 0;

	int warp_id = gid / 32;

	if (warp_id % 2 == 0)
	{
		a = 100.0;
		b = 50.0;
	}
	else
	{
		a = 200;
		b = 75;
	}
}

__global__ void divergence()
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	float a, b;
	a = b = 0;

	if (gid % 2 == 0)
	{
		a = 100.0;
		b = 50.0;
	}
	else
	{
		a = 200;
		b = 75;
	}
}

int main(int argc, char** argv)
{
	printf("\n-----------------------WARP DIVERGENCE EXAMPLE------------------------ \n\n");

	int size = 1 << 22;

	dim3 block_size(128);
	dim3 grid_size((size + block_size.x -1)/ block_size.x);

	without_divergence << <grid_size, block_size >> > ();
	hipDeviceSynchronize();

	divergence<< <grid_size, block_size >> > ();
	hipDeviceSynchronize();

	hipDeviceReset();
	return 0;
}