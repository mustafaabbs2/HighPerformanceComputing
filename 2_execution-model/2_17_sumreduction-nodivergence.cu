
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstdlib>
#include <iostream>
#include <numeric>
#include <vector>

using std::accumulate;
using std::cout;
using std::generate;
using std::vector;

#define SHMEM_SIZE 256

__global__ void sumReduction(int* v, int* v_r)
{
	// Allocate shared memory
	__shared__ int partial_sum[SHMEM_SIZE];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	partial_sum[threadIdx.x] = v[tid];
	__syncthreads();

	for(int s = 1; s < blockDim.x; s *= 2)
	{

		// This modulo operation is bad!! - there is an if condition causing warp divergence
		// if (threadIdx.x % (2 * s) == 0) {
		// 	partial_sum[threadIdx.x] += partial_sum[threadIdx.x + s];
		// }
		// instead .. do..

		int index = 2 * s * threadIdx.x;
		// when s = 1, for threadIdx.x = 0, index = 0, the first thread adds partial_sum[0] to partial_sum[1]
		// when s = 1, for threadIdx.x = 1, index = 2, the second thread adds partial_sum[2] to partial_sum[3]
		//.. and so on

		if(index < blockDim.x)
		{
			partial_sum[index] += partial_sum[index + s];
		}

		__syncthreads();
	}

	if(threadIdx.x == 0)
	{
		v_r[blockIdx.x] = partial_sum[0];
	}
}

int main()
{
	// Vector size
	int N = 1 << 16;
	size_t bytes = N * sizeof(int);

	// Host data
	vector<int> h_v(N);
	vector<int> h_v_r(N);

	// Initialize the input data
	generate(begin(h_v), end(h_v), []() { return rand() % 10; });

	// Allocate device memory
	int *d_v, *d_v_r;
	hipMalloc(&d_v, bytes);
	hipMalloc(&d_v_r, bytes);

	// Copy to device
	hipMemcpy(d_v, h_v.data(), bytes, hipMemcpyHostToDevice);

	// TB Size
	const int TB_SIZE = 256;

	int GRID_SIZE = N / TB_SIZE;

	sumReduction<<<GRID_SIZE, TB_SIZE>>>(d_v, d_v_r);
	sumReduction<<<1, TB_SIZE>>>(d_v_r, d_v_r);

	// Copy to host;
	hipMemcpy(h_v_r.data(), d_v_r, bytes, hipMemcpyDeviceToHost);

	// Print the result
	assert(h_v_r[0] == std::accumulate(begin(h_v), end(h_v), 0));

	cout << "COMPLETED SUCCESSFULLY\n";

	return 0;
	// nvcc -o 2_17_sumreduction-nodivergence 2_17_sumreduction-nodivergence.cu
}