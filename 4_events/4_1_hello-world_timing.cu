
#include <hip/hip_runtime.h>
#include<iostream>

__global__ void helloFromGPU(void)
{
    printf("Hello World from GPU! \n");
}

int main()
{   printf("Hello World from CPU! \n");

    float et;   
    hipEvent_t start, stop;
    hipEventCreate(&start); hipEventCreate(&stop);

//Start timing
    hipEventRecord(start);

    helloFromGPU <<<1,10>>>();

//Synchronize
    hipDeviceSynchronize();

//Stop
    hipEventRecord(stop);

//Sync events
    hipEventSynchronize(stop);

//Calculate et = elapsed time   
    hipEventElapsedTime(&et, start, stop);

    printf("The elapsed time is  %f milliseconds", et); 

    hipDeviceReset();
    return 0;
}