#include "hip/hip_runtime.h"

//C headers
#include <stdio.h>
#include <stdlib.h>
//need these CUDA headers
#include "hip/hip_runtime.h"
#include ""
//Contains self written helper functions
#include "../common/common.h"


__global__ void blocking_nonblocking_test1()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		for (size_t i = 0; i < 10000; i++)
		{
			printf("kernel 1 \n");
		}
	}
}

int main(int argc, char ** argv)
{
	int size = 1 << 15;
	
	hipStream_t stm1,stm2,stm3;
	CHECK(hipStreamCreateWithFlags(&stm1, hipStreamNonBlocking));
	CHECK(hipStreamCreate(&stm2));
	CHECK(hipStreamCreateWithFlags(&stm3,hipStreamNonBlocking));


	dim3 block(128);
	dim3 grid(size / block.x);

	blocking_nonblocking_test1 << <grid, block, 0 , stm1 >> > ();
	blocking_nonblocking_test1 << <grid, block, 0 , stm2>> > ();
	blocking_nonblocking_test1 << <grid, block, 0, stm3 >> > ();

	CHECK(hipStreamDestroy(stm1));
	CHECK(hipStreamDestroy(stm2));
	CHECK(hipStreamDestroy(stm3));
	CHECK(hipDeviceSynchronize());

	CHECK(hipDeviceReset());
	return 0;
}