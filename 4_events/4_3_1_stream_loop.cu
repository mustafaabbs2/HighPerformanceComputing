for (int i = 0; i < nStreams; i++) {
int offset = i * bytesPerStream;
hipMemcpyAsync(&d_a[offset], &a[offset], bytePerStream, streams[i]);
kernel<<grid, block, 0, streams[i]>>(&d_a[offset]);
hipMemcpyAsync(&a[offset], &d_a[offset], bytesPerStream, streams[i]);
}
for (int i = 0; i < nStreams; i++) {
hipStreamSynchronize(streams[i]);
}